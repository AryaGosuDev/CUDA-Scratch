#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <random>
#include <time.h>
#include <chrono>
#include <iostream>

#define checkCudaErrors(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) {\
        printf("Error : %s:%d, ", __FILE__, __LINE__); \
        printf("code:%d, reason : %s\n", error, hipGetErrorName(error)); \
        exit(-10 * error);\
    } \
} \

void initialData(float* ip, const int size) {
    for ( int i = 0; i < size; i++) ip[i] = (float)(rand() & 0xFF) / 10.0f;
}

void sumMatrixOnHost(float* A, float* B, float* C, const int nx, const int ny) {
    float* ia = A; float* ib = B; float* ic = C;

    for (int iy = 0; iy < ny; iy++) {
        for (int ix = 0; ix < nx; ix++) {
            ic[ix] = ia[ix] + ib[ix];
        }
        ia += nx; ib += nx; ic += nx;
    }
}

void checkResult(float* hostRef, float* gpuRef, const int N) {
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)  {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0;
            printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
            break;
        }
    }
    if (match) printf("Arrays match.\n\n"); else printf("Arrays do not match.\n\n");
}

__global__ void sumMatrixOnGPU2D(float* MatA, float* MatB, float* MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
        MatC[idx] = MatA[idx] + MatB[idx];
}

int main(int argc, char** argv)
{
    printf("%s Starting...\n", argv[0]);
    checkCudaErrors(hipSetDevice(0));

    // set up data size of matrix
    // elem size = 16,384
    int nx = 1 << 14;
    int ny = 1 << 14;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    float* h_A, * h_B, * hostRef, * gpuRef;
    h_A = (float*)malloc(nBytes);
    h_B = (float*)malloc(nBytes);
    hostRef = (float*)malloc(nBytes);
    gpuRef = (float*)malloc(nBytes);

    initialData(h_A, nxy);
    initialData(h_B, nxy);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);
    
    auto start = std::chrono::steady_clock::now();
    sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);
    auto end = std::chrono::steady_clock::now();
    auto diff = end - start;
    double elapsed = std::chrono::duration_cast<std::chrono::milliseconds> (diff).count();

    float* d_MatA, * d_MatB, * d_MatC;
    checkCudaErrors(hipMalloc((void**)&d_MatA, nBytes));
    checkCudaErrors(hipMalloc((void**)&d_MatB, nBytes));
    checkCudaErrors(hipMalloc((void**)&d_MatC, nBytes));

    checkCudaErrors(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice));

    int dimx = 16;
    int dimy = 16;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    sumMatrixOnGPU2D << <grid, block >> > (d_MatA, d_MatB, d_MatC, nx, ny);
    checkCudaErrors(hipDeviceSynchronize());
    printf("sumMatrixOnGPU2D <<<(%d,%d), (%d,%d)>>>\n", grid.x, grid.y, block.x, block.y);
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost));

    checkResult(hostRef, gpuRef, nxy);

    checkCudaErrors(hipFree(d_MatA));
    checkCudaErrors(hipFree(d_MatB));
    checkCudaErrors(hipFree(d_MatC));

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    checkCudaErrors(hipDeviceReset());

    return (0);
}
