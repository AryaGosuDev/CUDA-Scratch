#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <random>
#include <time.h>
#include <chrono>
#include <iostream>

using namespace std;

#define checkCudaErrors(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) {\
        printf("Error : %s:%d, ", __FILE__, __LINE__); \
        printf("code:%d, reason : %s\n", error, hipGetErrorName(error)); \
        exit(-10 * error);\
    } \
} \

#define BDIMX 16
#define BDIMY 16

void initialData(float* in, const int size)
{
    for (int i = 0; i < size; i++)
    {
        in[i] = (float)(rand() & 0xFF) / 10.0f; //100.0f;
    }

    return;
}

void printData(float* in, const int size)
{
    for (int i = 0; i < size; i++)
    {
        printf("%dth element: %f\n", i, in[i]);
    }

    return;
}

void checkCudaErrorsResult(float* hostRef, float* gpuRef, const int size, int showme)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < size; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("different on %dth element: host %f gpu %f\n", i, hostRef[i],
                gpuRef[i]);
            break;
        }

        if (showme && i > size / 2 && i < size / 2 + 5)
        {
            // printf("%dth element: host %f gpu %f\n",i,hostRef[i],gpuRef[i]);
        }
    }

    if (!match)  printf("Arrays do not match.\n\n");
}

void transposeHost(float* out, float* in, const int nx, const int ny)
{
    for (int iy = 0; iy < ny; ++iy)
    {
        for (int ix = 0; ix < nx; ++ix)
        {
            out[ix * ny + iy] = in[iy * nx + ix];
        }
    }
}

__global__ void warmup(float* out, float* in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[iy * nx + ix];
    }
}

// case 0 copy kernel: access data in rows
__global__ void copyRow(float* out, float* in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[iy * nx + ix];
    }
}
 
// case 1 copy kernel: access data in columns
__global__ void copyCol(float* out, float* in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[ix * ny + iy] = in[ix * ny + iy];
    }
}
     
// case 2 transpose kernel: read in rows and write in columns
__global__ void transposeNaiveRow(float* out, float* in, const int nx,
    const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}

// case 3 transpose kernel: read in columns and write in rows
__global__ void transposeNaiveCol(float* out, float* in, const int nx,
    const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}

// case 4 transpose kernel: read in rows and write in columns + unroll 4 blocks
__global__ void transposeUnroll4Row(float* out, float* in, const int nx,
    const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        out[to] = in[ti];
        out[to + ny * blockDim.x] = in[ti + blockDim.x];
        out[to + ny * 2 * blockDim.x] = in[ti + 2 * blockDim.x];
        out[to + ny * 3 * blockDim.x] = in[ti + 3 * blockDim.x];
    }
}

// case 5 transpose kernel: read in columns and write in rows + unroll 4 blocks
__global__ void transposeUnroll4Col(float* out, float* in, const int nx,
    const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        out[ti] = in[to];
        out[ti + blockDim.x] = in[to + blockDim.x * ny];
        out[ti + 2 * blockDim.x] = in[to + 2 * blockDim.x * ny];
        out[ti + 3 * blockDim.x] = in[to + 3 * blockDim.x * ny];
    }
}

/*
 * case 6 :  transpose kernel: read in rows and write in colunms + diagonal
 * coordinate transform
 */
__global__ void transposeDiagonalRow(float* out, float* in, const int nx,
    const int ny)
{
    unsigned int blk_y = blockIdx.x;
    unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}

/*
 * case 7 :  transpose kernel: read in columns and write in row + diagonal
 * coordinate transform.
 */
__global__ void transposeDiagonalCol(float* out, float* in, const int nx,
    const int ny)
{
    unsigned int blk_y = blockIdx.x;
    unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}

 

//profile with both L1 cache turned off and off : to turn off l1 load cache, compile with flag  --ptxas-options=-dlcm=cg
int main(int argc, char** argv)
{
    printf("%s Starting...\n", argv[0]);
    checkCudaErrors(hipSetDevice(0));

    // set up array size 2048
    int nx = 1 << 11;
    int ny = 1 << 11;

    int iKernel = 0;
    int blockx = 16;
    int blocky = 16;

    if (argc > 1) iKernel = atoi(argv[1]);

    if (argc > 2) blockx = atoi(argv[2]);

    if (argc > 3) blocky = atoi(argv[3]);

    if (argc > 4) nx = atoi(argv[4]);

    if (argc > 5) ny = atoi(argv[5]);

    printf(" with matrix nx %d ny %d with kernel %d\n", nx, ny, iKernel);
    size_t nBytes = nx * ny * sizeof(float);

    // execution configuration
    dim3 block(blockx, blocky);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // allocate host memory
    float* h_A = (float*)malloc(nBytes);
    float* hostRef = (float*)malloc(nBytes);
    float* gpuRef = (float*)malloc(nBytes);

    // initialize host array
    initialData(h_A, nx * ny);

    // transpose at host side
    transposeHost(hostRef, h_A, nx, ny);

    // allocate device memory
    float* d_A, * d_C;
    checkCudaErrors(hipMalloc((float**)&d_A, nBytes));
    checkCudaErrors(hipMalloc((float**)&d_C, nBytes));

    // copy data from host to device
    checkCudaErrors(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));

    // warmup to avoide startup overhead
    auto start = std::chrono::steady_clock::now();
    warmup << <grid, block >> > (d_C, d_A, nx, ny);
    checkCudaErrors(hipDeviceSynchronize());
    auto end = std::chrono::steady_clock::now(); auto diff = end - start;
    double elapsed = std::chrono::duration<double>(diff).count();
    printf("warmup         elapsed %f sec\n", elapsed);
    checkCudaErrors(hipGetLastError());

    vector < pair<void (*)(float*, float*, int, int), string> > kernelArray;
     
    kernelArray.push_back({ &copyRow, "CopyRow       " });
    kernelArray.push_back({ &copyCol, "CopyCol       " });
    kernelArray.push_back({ &transposeNaiveRow, "NaiveRow       " });
    kernelArray.push_back({ &transposeNaiveCol, "NaiveCol       " });
    kernelArray.push_back({ &transposeUnroll4Row, "Unroll4Row       " });
    kernelArray.push_back({ &transposeUnroll4Col, "Unroll4Col       " });
    kernelArray.push_back({ &transposeDiagonalRow, "DiagonalRow       " });
    kernelArray.push_back({ &transposeDiagonalCol, "DiagonalCol       " });

    for (int iKernel = 0; iKernel < kernelArray.size(); ++iKernel) {
        // execution configuration
        dim3 block(blockx, blocky);
        dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

        if (iKernel == 4 || iKernel == 5) grid.x = (nx + block.x * 4 - 1) / (block.x * 4);

        start = std::chrono::steady_clock::now();
        kernelArray[iKernel].first << <grid, block >> > (d_C, d_A, nx, ny);
        checkCudaErrors(hipDeviceSynchronize());
        end = std::chrono::steady_clock::now(); diff = end - start;
        elapsed = std::chrono::duration<double>(diff).count();

        // calculate effective_bandwidth
        float ibnd = 2 * nx * ny * sizeof(float) / 1e9 / elapsed;
        printf("%s elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>> effective "
            "bandwidth %f GB\n", (kernelArray[iKernel].second).c_str(), elapsed, grid.x, grid.y, block.x,
            block.y, ibnd);
        checkCudaErrors(hipGetLastError());

        // checkCudaErrors kernel results
        if (iKernel > 1)
        {
            checkCudaErrors(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
            checkCudaErrorsResult(hostRef, gpuRef, nx * ny, 1);
        }
    }


    // free host and device memory
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_C));
    free(h_A);
    free(hostRef);
    free(gpuRef);

    // reset device
    checkCudaErrors(hipDeviceReset());
    return EXIT_SUCCESS;
}