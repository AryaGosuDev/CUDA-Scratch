#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <string>
#include <Windows.h>
#include <cstdlib> 

#define checkCudaErrors(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) {\
        printf("Error : %s:%d, ", __FILE__, __LINE__); \
        printf("code:%d, reason : %s\n", error, hipGetErrorName(error)); \
        exit(-10 * error);\
    } \
} \

#define NSTREAM 4
#define BDIM 128

void initialData(float* ip, int size)
{
    int i;

    for (i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void sumArraysOnHost(float* A, float* B, float* C, const int N)
{
    for (int idx = 0; idx < N; idx++)
        C[idx] = A[idx] + B[idx];
}

__global__ void sumArrays(float* A, float* B, float* C, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N)
    {
        for (int i = 0; i < N; ++i)
        {
            C[idx] = A[idx] + B[idx];
        }
    }
}

void checkResult(float* hostRef, float* gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match.\n\n");
}


int main(int argc, char** argv)
{
    printf("> %s Starting...\n", argv[0]);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s\n", dev, deviceProp.name);
    checkCudaErrors(hipSetDevice(dev));

    // check if device support hyper-q
    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5))
    {
        if (deviceProp.concurrentKernels == 0)
        {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 "
                "or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else
        {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n",
        deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);



    // set up max connection
    SetEnvironmentVariable("CUDA_DEVICE_MAX_CONNECTIONS", "1");
    char buffer[100];
    DWORD bufferSize = sizeof(buffer) / sizeof(char);
    GetEnvironmentVariable("CUDA_DEVICE_MAX_CONNECTIONS", buffer, bufferSize);
    printf("%s = %s\n", "CUDA_DEVICE_MAX_CONNECTIONS", buffer);
    printf("> with streams = %d\n", NSTREAM);

    // set up data size of vectors
    int nElem = 1 << 18;
    printf("> vector size = %d\n", nElem);
    size_t nBytes = nElem * sizeof(float);

    // malloc pinned host memory for async memcpy
    float* h_A, * h_B, * hostRef, * gpuRef;
    checkCudaErrors(hipHostAlloc((void**)&h_A, nBytes, hipHostMallocDefault));
    checkCudaErrors(hipHostAlloc((void**)&h_B, nBytes, hipHostMallocDefault));
    checkCudaErrors(hipHostAlloc((void**)&gpuRef, nBytes, hipHostMallocDefault));
    checkCudaErrors(hipHostAlloc((void**)&hostRef, nBytes, hipHostMallocDefault));

    // initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add vector at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    // malloc device global memory
    float* d_A, * d_B, * d_C;
    checkCudaErrors(hipMalloc((float**)&d_A, nBytes));
    checkCudaErrors(hipMalloc((float**)&d_B, nBytes));
    checkCudaErrors(hipMalloc((float**)&d_C, nBytes));

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // invoke kernel at host side
    dim3 block(BDIM);
    dim3 grid((nElem + block.x - 1) / block.x);
    printf("> grid (%d, %d) block (%d, %d)\n", grid.x, grid.y, block.x,
        block.y);

    // sequential operation
    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    float memcpy_h2d_time;
    checkCudaErrors(hipEventElapsedTime(&memcpy_h2d_time, start, stop));

    checkCudaErrors(hipEventRecord(start, 0));
    sumArrays << <grid, block >> > (d_A, d_B, d_C, nElem);
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    float kernel_time;
    checkCudaErrors(hipEventElapsedTime(&kernel_time, start, stop));

    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    float memcpy_d2h_time;
    checkCudaErrors(hipEventElapsedTime(&memcpy_d2h_time, start, stop));
    float itotal = kernel_time + memcpy_h2d_time + memcpy_d2h_time;

    printf("\n");
    printf("Measured timings (throughput):\n");
    printf(" Memcpy host to device\t: %f ms (%f GB/s)\n",
        memcpy_h2d_time, (nBytes * 1e-6) / memcpy_h2d_time);
    printf(" Memcpy device to host\t: %f ms (%f GB/s)\n",
        memcpy_d2h_time, (nBytes * 1e-6) / memcpy_d2h_time);
    printf(" Kernel\t\t\t: %f ms (%f GB/s)\n",
        kernel_time, (nBytes * 2e-6) / kernel_time);
    printf(" Total\t\t\t: %f ms (%f GB/s)\n",
        itotal, (nBytes * 2e-6) / itotal);

    // grid parallel operation
    int iElem = nElem / NSTREAM;
    size_t iBytes = iElem * sizeof(float);
    grid.x = (iElem + block.x - 1) / block.x;

    hipStream_t stream[NSTREAM];

    for (int i = 0; i < NSTREAM; ++i)
    {
        checkCudaErrors(hipStreamCreate(&stream[i]));
    }

    checkCudaErrors(hipEventRecord(start, 0));

    // initiate all work on the device asynchronously in depth-first order
    for (int i = 0; i < NSTREAM; ++i)
    {
        int ioffset = i * iElem;
        checkCudaErrors(hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes,
            hipMemcpyHostToDevice, stream[i]));
        checkCudaErrors(hipMemcpyAsync(&d_B[ioffset], &h_B[ioffset], iBytes,
            hipMemcpyHostToDevice, stream[i]));
        sumArrays << <grid, block, 0, stream[i] >> > (&d_A[ioffset], &d_B[ioffset],
            &d_C[ioffset], iElem);
        checkCudaErrors(hipMemcpyAsync(&gpuRef[ioffset], &d_C[ioffset], iBytes,
            hipMemcpyDeviceToHost, stream[i]));
    }

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    float execution_time;
    checkCudaErrors(hipEventElapsedTime(&execution_time, start, stop));

    printf("\n");
    printf("Actual results from overlapped data transfers:\n");
    printf(" overlap with %d streams : %f ms (%f GB/s)\n", NSTREAM,
        execution_time, (nBytes * 2e-6) / execution_time);
    printf(" speedup                : %f \n",
        ((itotal - execution_time) * 100.0f) / itotal);

    // check kernel error
    checkCudaErrors(hipGetLastError());

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    // free host memory
    checkCudaErrors(hipHostFree(h_A));
    checkCudaErrors(hipHostFree(h_B));
    checkCudaErrors(hipHostFree(hostRef));
    checkCudaErrors(hipHostFree(gpuRef));

    // destroy events
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    // destroy streams
    for (int i = 0; i < NSTREAM; ++i)
    {
        checkCudaErrors(hipStreamDestroy(stream[i]));
    }

    checkCudaErrors(hipDeviceReset());
    return(0);
}