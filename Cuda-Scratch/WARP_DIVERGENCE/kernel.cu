#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <random>
#include <time.h>
#include <chrono>
#include <iostream>


#define checkCudaErrors(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) {\
        printf("Error : %s:%d, ", __FILE__, __LINE__); \
        printf("code:%d, reason : %s\n", error, hipGetErrorName(error)); \
        exit(-10 * error);\
    } \
} \

/*
 * simpleDivergence demonstrates divergent code on the GPU and its impact on
 * performance and CUDA metrics.
 */

__global__ void mathKernel1(float* c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if (tid % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void mathKernel2(float* c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if ((tid / warpSize) % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void mathKernel3(float* c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    bool ipred = (tid % 2 == 0);

    if (ipred)
    {
        ia = 100.0f;
    }

    if (!ipred)
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void mathKernel4(float* c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    int itid = tid >> 5;

    if (itid & 0x01 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void warmingup(float* c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if ((tid / warpSize) % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}


int main(int argc, char** argv)
{
    printf("Starting...\n");
    checkCudaErrors(hipSetDevice(0));

    // set up data size
    int size = 64;
    int blocksize = 64;

    if (argc > 1) blocksize = atoi(argv[1]);

    if (argc > 2) size = atoi(argv[2]);

    printf("Data size %d ", size);

    // set up execution configuration
    dim3 block(blocksize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    printf("Execution Configure (block %d grid %d)\n", block.x, grid.x);

    // allocate gpu memory
    float* d_C;
    size_t nBytes = size * sizeof(float);
    checkCudaErrors(hipMalloc((float**)&d_C, nBytes));

    // run a warmup kernel to remove overhead
    checkCudaErrors(hipDeviceSynchronize());
    warmingup << <grid, block >> > (d_C);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    // run kernel 1
    mathKernel1 << <grid, block >> > (d_C);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    // run kernel 3
    mathKernel2 << <grid, block >> > (d_C);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    // run kernel 3
    mathKernel3 << <grid, block >> > (d_C);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    // run kernel 4
    mathKernel4 << <grid, block >> > (d_C);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    // free gpu memory and reset divece
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipDeviceReset());
    return EXIT_SUCCESS;
}