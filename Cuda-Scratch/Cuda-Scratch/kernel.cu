﻿
#include "hip/hip_runtime.h"


#include <stdio.h>


#define checkCudaErrors(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) {\
        printf("Error : %s:%d, ", __FILE__, __LINE__); \
        printf("code:%d, reason : %s\n", error, hipGetErrorName(error)); \
        exit(-10 * error);\
    } \
} \

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    checkCudaErrors(addWithCuda(c, a, b, arraySize));
    
    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    checkCudaErrors(hipDeviceReset());
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    checkCudaErrors(hipSetDevice(0));

    // Allocate GPU buffers for three vectors (two input, one output)    .
    checkCudaErrors(hipMalloc((void**)&dev_c, size * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&dev_a, size * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&dev_b, size * sizeof(int)));

    // Copy input vectors from host memory to GPU buffers.
    checkCudaErrors(hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice));
    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
    // Check for any errors launching the kernel
    checkCudaErrors(hipGetLastError());
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    checkCudaErrors(hipDeviceSynchronize());

    // Copy output vector from GPU buffer to host memory.
    checkCudaErrors(hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost));

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return hipSuccess;
}
