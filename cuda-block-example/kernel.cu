#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "../cuda_include.h"

#include <stdio.h>

/*
 * This example helps to visualize the relationship between thread/block IDs and
 * offsets into data. For each CUDA thread, this example displays the
 * intra-block thread ID, the inter-block block ID, the global coordinate of a
 * thread, the calculated offset into input data, and the input data at that
 * offset.
 */

void printMatrix(int* C, const int nx, const int ny)
{
    int* ic = C;
    printf("\nMatrix: (%d.%d)\n", nx, ny);

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            printf("%3d", ic[ix]);

        }

        ic += nx;
        printf("\n");
    }

    printf("\n");
    return;
}

__global__ void printThreadIndex(int* A, const int nx, const int ny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    printf("thread_id (%d,%d) block_id (%d,%d) coordinate (%d,%d) global index"
        " %2d ival %2d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y,
        ix, iy, idx, A[idx]);
}


void gridBlockSizes() {
    // define total data element
    int nElem = 1024;

    // define grid and block structure
    dim3 block(1024);
    dim3 grid((nElem + block.x - 1) / block.x);
    printf("grid.x %d block.x %d \n", grid.x, block.x);

    // reset block
    block.x = 512;
    grid.x = (nElem + block.x - 1) / block.x;
    printf("grid.x %d block.x %d \n", grid.x, block.x);

    // reset block
    block.x = 256;
    grid.x = (nElem + block.x - 1) / block.x;
    printf("grid.x %d block.x %d \n", grid.x, block.x);

    // reset block
    block.x = 128;
    grid.x = (nElem + block.x - 1) / block.x;
    printf("grid.x %d block.x %d \n", grid.x, block.x);

    // reset device before you leave
    //CHECK(hipDeviceReset());
}


int main(int argc, char** argv)
{
    printf("%s Starting...\n", argv[0]);

    // get device information
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
    gridBlockSizes();

    // set matrix dimension
    int nx = 8;
    int ny = 6;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);

    // malloc host memory
    int* h_A;
    h_A = (int*)malloc(nBytes);

    // iniitialize host matrix with integer
    for (int i = 0; i < nxy; i++)
    {
        h_A[i] = i;
    }
    printMatrix(h_A, nx, ny);

    // malloc device memory
    int* d_MatA;
    CHECK(hipMalloc((void**)&d_MatA, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));

    // set up execution configuration
    dim3 block(4, 2);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // invoke the kernel
    printThreadIndex << <grid, block >> > (d_MatA, nx, ny);
    CHECK(hipGetLastError());

    // free host and devide memory
    CHECK(hipFree(d_MatA));
    free(h_A);

    // reset device
    CHECK(hipDeviceReset());

    return (0);
}